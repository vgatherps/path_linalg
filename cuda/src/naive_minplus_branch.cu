#include "hip/hip_runtime.h"
#include "types.hh"

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

/*
Matrix sizes:
MxK * KxN = MxN
*/

// We assume that A, B, and C are stored in row-major order

constexpr static int NUM_ROWS_BRANCHLESS = 50;

// For the first 50 rows of the matrix, go branchless as we assume there will be many overwrites
// however assume that by 50 rows, we'll have enough points to be close th a true minimum
// and can skip over the rest
__global__ static void naive_minplus_branch_cu(int M, int N, int K,
                                               const float *A_cost, const float *B_cost, float *C_cost,
                                               const uint *A_prime, const uint *B_prime, uint *C_prime)
{
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if (K <= 0)
    {
        return;
    }

    if (x < M && y < N)
    {
        float min_cost = A_cost[x * K] + B_cost[y];
        uint min_prime = A_prime[x * K] * B_prime[y];
        int i = 1;
        int min_k_branchless = K < NUM_ROWS_BRANCHLESS ? K : NUM_ROWS_BRANCHLESS;
        for (; i < min_k_branchless; ++i)
        {

            float i_cost = A_cost[x * K + i] + B_cost[i * N + y];
            uint i_prime = A_prime[x * K + i] * B_prime[i * N + y];

            // This is converted into predicates by the compiler
            if (i_cost < min_cost)
            {
                min_cost = i_cost;
                min_prime = i_prime;
            }
        }
        for (; i < K; ++i)
        {

            // This is structured to force a branch
            // assumption is that after some traversal we'll have found a minimum
            // and we avoid memory bandwidth of the primes matrix

            float i_cost = A_cost[x * K + i] + B_cost[i * N + y];

            if (i_cost < min_cost)
            {
                // As this load is inside the branch, the compiler
                // does not hoist it into a predicate
                uint i_prime = A_prime[x * K + i] * B_prime[i * N + y];
                min_cost = i_cost;
                min_prime = i_prime;
            }
        }
        C_cost[x * N + y] = min_cost;
        C_prime[x * N + y] = min_prime;
    }
}

void naive_minplus_branch(int M, int N, int K,
                          const float *A_cost, const float *B_cost, float *C_cost,
                          const uint *A_prime, const uint *B_prime, uint *C_prime)
{
    dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
    dim3 blockDim(32, 32);
    naive_minplus_branch_cu<<<gridDim, blockDim>>>(M, N, K, A_cost, B_cost, C_cost, A_prime, B_prime, C_prime);
}